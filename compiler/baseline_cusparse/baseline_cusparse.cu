#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2022 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO LICENSEE:
 *
 * This source code and/or documentation ("Licensed Deliverables") are
 * subject to NVIDIA intellectual property rights under U.S. and
 * international Copyright laws.
 *
 * These Licensed Deliverables contained herein is PROPRIETARY and
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and
 * conditions of a form of NVIDIA software license agreement by and
 * between NVIDIA and Licensee ("License Agreement") or electronically
 * accepted by Licensee.  Notwithstanding any terms or conditions to
 * the contrary in the License Agreement, reproduction or disclosure
 * of the Licensed Deliverables to any third party without the express
 * written consent of NVIDIA is prohibited.
 *
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, NVIDIA MAKES NO REPRESENTATION ABOUT THE
 * SUITABILITY OF THESE LICENSED DELIVERABLES FOR ANY PURPOSE.  IT IS
 * PROVIDED "AS IS" WITHOUT EXPRESS OR IMPLIED WARRANTY OF ANY KIND.
 * NVIDIA DISCLAIMS ALL WARRANTIES WITH REGARD TO THESE LICENSED
 * DELIVERABLES, INCLUDING ALL IMPLIED WARRANTIES OF MERCHANTABILITY,
 * NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * NOTWITHSTANDING ANY TERMS OR CONDITIONS TO THE CONTRARY IN THE
 * LICENSE AGREEMENT, IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY
 * SPECIAL, INDIRECT, INCIDENTAL, OR CONSEQUENTIAL DAMAGES, OR ANY
 * DAMAGES WHATSOEVER RESULTING FROM LOSS OF USE, DATA OR PROFITS,
 * WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE OR OTHER TORTIOUS
 * ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE OR PERFORMANCE
 * OF THESE LICENSED DELIVERABLES.
 *
 * U.S. Government End Users.  These Licensed Deliverables are a
 * "commercial item" as that term is defined at 48 C.F.R. 2.101 (OCT
 * 1995), consisting of "commercial computer software" and "commercial
 * computer software documentation" as such terms are used in 48
 * C.F.R. 12.212 (SEPT 1995) and is provided to the U.S. Government
 * only as a commercial end item.  Consistent with 48 C.F.R.12.212 and
 * 48 C.F.R. 227.7202-1 through 227.7202-4 (JUNE 1995), all
 * U.S. Government End Users acquire the Licensed Deliverables with
 * only those rights set forth herein.
 *
 * Any use of the Licensed Deliverables in individual and commercial
 * software must include, in the user documentation and internal
 * comments to the code, the above Disclaimer and U.S. Government End
 * Users Notice.
 */
#include <hip/hip_runtime_api.h> // hipMalloc, hipMemcpy, etc.
#include <hipsparse.h>         // cusparseSpGEMM
#include <stdio.h>            // printf
#include <math.h>            // fabs
#include <stdlib.h>           // EXIT_FAILURE
#include <taco.h>
#include <taco/storage/file_io_mtx.h>
#include <taco/format.h>
#include <taco/index_notation/transformations.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/host_vector.h>
#include "benchmark/benchmark.h"
#include "../bench.h"
#include <tuple>
// #include "benchmark/include/benchmark/benchmark.h"

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

bool float_compare(float f1, float f2, float pct){
    if((f1 == 0.0f) && (f2 == 0.0f)){
        return true;
    }
    auto div_by = f2;
    if(f2 == 0.0f){
        div_by = f1;
    }
    auto percent_diff = (fabs(f1 - f2) / div_by) * 100.0;
    return percent_diff < pct;
}

int spgemm(taco::Tensor<float> & tensorA, taco::Tensor<float> & tensorB, benchmark::State * state){

    bool benching = (state != nullptr);

    if(benching) {state->PauseTiming();}

    std::cout << "Calculating SPGEMM" << std::endl;
    auto dims = tensorA.getDimensions();

    // Get A and its arrays to analyze sizes/etc
    int * rowptrA;
    int * colidxA;
    float * valsA;

    taco::getCSRArrays(tensorA, &rowptrA, &colidxA, &valsA);

    auto storage = tensorA.getStorage();
    auto index = storage.getIndex();
    auto rowptrArr = index.getModeIndex(1).getIndexArray(0);
    auto colidxArr = index.getModeIndex(1).getIndexArray(1);
    auto rowptrsize = rowptrArr.getSize();
    auto numrowsA = rowptrsize - 1;
    auto coloffsetsize = colidxArr.getSize();
    auto valssize = coloffsetsize;

    int NUM_I = dims[0];
    int NUM_K = dims[1];
    int NUM_J = dims[0];

    // tensorB is transposed in CSR
    taco::Tensor<float> tensorB_csc = tensorB.transpose("tensorB_transposed_csc", {0, 1}, taco::CSC);

    int * rowptrB;
    int * colidxB;
    float * valsB;

    taco::getCSRArrays(tensorB, &rowptrB, &colidxB, &valsB);

    /*
        Compute the output of spgemm
    */
    taco::IndexVar i, j, k;

    taco::Tensor<float> expected("expected", {NUM_I, NUM_J}, taco::CSR);
    expected(i, k) = tensorA(i, j) * tensorB_csc(j, k);
    expected.compile();
    expected.assemble();
    expected.compute();

    // Remove 0's or else the GPU stuff doesn't work due to mismatches in allocs.
    // auto expected_no_zeros = expected.removeExplicitZeros(taco::CSR);
    auto expected_no_zeros = expected.removeExplicitZeros(taco::CSR);

    // Get the information/ptrs from C for comparison and GPU calls
    int * rowptrC;
    int * colidxC;
    float * valsC;

    taco::getCSRArrays(expected_no_zeros, &rowptrC, &colidxC, &valsC);

    auto storageC = expected_no_zeros.getStorage();
    auto indexC = storageC.getIndex();

    auto rowptrArrC = indexC.getModeIndex(1).getIndexArray(0);
    auto colidxArrC = indexC.getModeIndex(1).getIndexArray(1);

    auto rowptrsizeC = rowptrArrC.getSize();
    auto coloffsetsizeC = colidxArrC.getSize();
    auto valssizeC = coloffsetsizeC;

    // Host problem definition
    #define   A_NUM_ROWS dims[0]   // C compatibility
    const int A_num_rows = dims[0];
    const int A_num_cols = dims[1];
    const int A_nnz      = valssize;
    const int B_num_rows = dims[1];
    const int B_num_cols = dims[0];
    const int B_nnz      = valssize;

    int  * hA_csrOffsets = rowptrA;
    int  * hA_columns    = colidxA;
    float* hA_values     = valsA;
    int  * hB_csrOffsets = rowptrB;
    int  * hB_columns    = colidxB;
    float* hB_values     = valsB;
    int   *hC_csrOffsets = rowptrC;
    int   *hC_columns    = colidxC;
    float *hC_values     = valsC;

    const int C_nnz       = valssizeC;
    #define   C_NUM_NNZ valssizeC   // C compatibility
    float               alpha       = 1.0f;
    float               beta        = 0.0f;
    hipsparseOperation_t opA         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipDataType        computeType = HIP_R_32F;
    //--------------------------------------------------------------------------
    // Device memory management: Allocate and copy A, B
    int   *dA_csrOffsets, *dA_columns, *dB_csrOffsets, *dB_columns,
        *dC_csrOffsets, *dC_columns;
    float *dA_values, *dB_values, *dC_values;
    // allocate A
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                        (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, A_nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dA_values,  A_nnz * sizeof(float)) )
    // allocate B
    CHECK_CUDA( hipMalloc((void**) &dB_csrOffsets,
                        (B_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_columns, B_nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dB_values,  B_nnz * sizeof(float)) )
    // allocate C offsets
    CHECK_CUDA( hipMalloc((void**) &dC_csrOffsets,
                        (A_num_rows + 1) * sizeof(int)) )
    // copy A
    CHECK_CUDA( hipMemcpy(dA_csrOffsets, hA_csrOffsets,
    // CHECK_CUDA( hipMemcpy(dA_csrOffsets, &hA_csrOffsets,
                        (A_num_rows + 1) * sizeof(int),
                        hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int),
                        hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, hA_values,
                        A_nnz * sizeof(float), hipMemcpyHostToDevice) )
    // copy B
    CHECK_CUDA( hipMemcpy(dB_csrOffsets, hB_csrOffsets,
                        (B_num_rows + 1) * sizeof(int),
                        hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB_columns, hB_columns, B_nnz * sizeof(int),
                        hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB_values, hB_values,
                        B_nnz * sizeof(float), hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA, matB, matC;
    void*  dBuffer1    = NULL, *dBuffer2   = NULL;
    size_t bufferSize1 = 0,    bufferSize2 = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                    dA_csrOffsets, dA_columns, dA_values,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matB, B_num_rows, B_num_cols, B_nnz,
                                    dB_csrOffsets, dB_columns, dB_values,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matC, A_num_rows, B_num_cols, 0,
                                    dC_csrOffsets, NULL, NULL,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    //--------------------------------------------------------------------------
    // SpGEMM Computation
    hipsparseSpGEMMDescr_t spgemmDesc;
    CHECK_CUSPARSE( hipsparseSpGEMM_createDescr(&spgemmDesc) )

    if(benching) {state->ResumeTiming();}

    // ask bufferSize1 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                    &alpha, matA, matB, &beta, matC,
                                    computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                    spgemmDesc, &bufferSize1, NULL) )

    CHECK_CUDA( hipMalloc((void**) &dBuffer1, bufferSize1) )
    // inspect the matrices A and B to understand the memory requirement for
    // the next step
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                    &alpha, matA, matB, &beta, matC,
                                    computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                    spgemmDesc, &bufferSize1, dBuffer1) )

    // ask bufferSize2 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_compute(handle, opA, opB,
                            &alpha, matA, matB, &beta, matC,
                            computeType, HIPSPARSE_SPGEMM_DEFAULT,
                            spgemmDesc, &bufferSize2, NULL) )
    CHECK_CUDA( hipMalloc((void**) &dBuffer2, bufferSize2) )

    // compute the intermediate product of A * B
    CHECK_CUSPARSE( hipsparseSpGEMM_compute(handle, opA, opB,
                                        &alpha, matA, matB, &beta, matC,
                                        computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                        spgemmDesc, &bufferSize2, dBuffer2) )
    // get matrix C non-zero entries C_nnz1
    int64_t C_num_rows1, C_num_cols1, C_nnz1;
    CHECK_CUSPARSE( hipsparseSpMatGetSize(matC, &C_num_rows1, &C_num_cols1,
                                        &C_nnz1) )
    // allocate matrix C
    CHECK_CUDA( hipMalloc((void**) &dC_columns, C_nnz1 * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dC_values,  C_nnz1 * sizeof(float)) )

    // NOTE: if 'beta' != 0, the values of C must be update after the allocation
    //       of dC_values, and before the call of hipsparseSpGEMM_copy

    // update matC with the new pointers
    CHECK_CUSPARSE(
        hipsparseCsrSetPointers(matC, dC_csrOffsets, dC_columns, dC_values) )

    // if beta != 0, hipsparseSpGEMM_copy reuses/updates the values of dC_values

    // copy the final products to the matrix C
    CHECK_CUSPARSE(
        hipsparseSpGEMM_copy(handle, opA, opB,
                            &alpha, matA, matB, &beta, matC,
                            computeType, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc) )

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseSpGEMM_destroyDescr(spgemmDesc) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    // device result check
    int   hC_csrOffsets_tmp[A_NUM_ROWS + 1];
    int   hC_columns_tmp[C_nnz1];
    float hC_values_tmp[C_nnz1];
    CHECK_CUDA( hipMemcpy(hC_csrOffsets_tmp, dC_csrOffsets,
                        (A_num_rows + 1) * sizeof(int),
                        hipMemcpyDeviceToHost) )
    CHECK_CUDA( hipMemcpy(hC_columns_tmp, dC_columns, C_nnz1 * sizeof(int),
                        hipMemcpyDeviceToHost) )
    CHECK_CUDA( hipMemcpy(hC_values_tmp, dC_values, C_nnz1 * sizeof(float),
                        hipMemcpyDeviceToHost) )

    if(benching) {state->PauseTiming();}

    taco::Tensor<float> out_gpu = taco::makeCSR("output_from_gpu", {dims[0], dims[0]}, &hC_csrOffsets_tmp[0], &hC_columns_tmp[0], &hC_values_tmp[0]);

    // Need to squeeze out some zeros
    auto out_gpu_no_zeros = out_gpu.removeExplicitZeros(taco::CSR);

    int * rowptr_out;
    int * colidx_out;
    float * vals_out;

    taco::getCSRArrays(out_gpu_no_zeros, &rowptr_out, &colidx_out, &vals_out);

    int correct = 1;
    for (int i = 0; i < A_num_rows + 1; i++) {
        if (rowptr_out[i] != hC_csrOffsets[i]) {
            std::cout << "ROWS GPU: " << rowptr_out[i] << " COMPARED TO CPU: " << hC_csrOffsets[i] << std::endl;
            correct = 0;
            break;
        }
    }

    for (int i = 0; i < C_nnz1; i++) {
        if (colidx_out[i] != hC_columns[i]){
            std::cout << "COL GPU: " << colidx_out[i] << " COMPARED TO CPU: " << hC_columns[i] << std::endl;
            correct = 0;
            break;
        }
        else if(!float_compare(vals_out[i], hC_values[i], 0.01f)) { // direct floating point
            std::cout << "VAL GPU: " << vals_out[i] << " COMPARED TO CPU: " << hC_values[i] << std::endl;
            correct = 0;                         // comparison is not reliable
            break;
        }
    }
    if (correct)
        printf("spgemm_example test PASSED\n");
    else {
        printf("spgemm_example test FAILED: wrong result\n");
        return EXIT_FAILURE;
    }
    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer1) )
    CHECK_CUDA( hipFree(dBuffer2) )
    CHECK_CUDA( hipFree(dA_csrOffsets) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    CHECK_CUDA( hipFree(dB_csrOffsets) )
    CHECK_CUDA( hipFree(dB_columns) )
    CHECK_CUDA( hipFree(dB_values) )
    CHECK_CUDA( hipFree(dC_csrOffsets) )
    CHECK_CUDA( hipFree(dC_columns) )
    CHECK_CUDA( hipFree(dC_values) )
    return EXIT_SUCCESS;
}

int sddmm(std::string mat_path){

    std::cout << "Calculating SDDMM" << std::endl;
    taco::TensorBase tb_denseA_double = taco::readMTX(mat_path, {taco::Dense, taco::Dense});
    auto dims = tb_denseA_double.getDimensions();

    //  Have double A, need to convert to float first...
    auto storage_A_double_arr = tb_denseA_double.getStorage().getValues();
    auto valsAsize = storage_A_double_arr.getSize();
    auto valsA_double = (double *) storage_A_double_arr.getData();
    float * valsA = (float *) malloc(valsAsize * sizeof(float));
    for(int i_ = 0; i_ < valsAsize; i_++){
        valsA[i_] = (float) valsA_double[i_];
    }

    taco::Tensor<float> tb_denseA(dims, {taco::Dense, taco::Dense}, 0);
    tb_denseA.setName("tb_denseA");
    for(int x = 0; x < dims[0]; x++){
        for(int y = 0; y < dims[1]; y++){
            tb_denseA.insert({x, y}, valsA[x * dims[1] + y]);
        }
    }
    taco::Tensor<float> tb_denseB = tb_denseA.transpose("tb_denseB", {1, 0}, {taco::Dense, taco::Dense});
    auto valsB = (float *) tb_denseB.getStorage().getValues().getData();

    // Compute C as A*B so dimensions match
    taco::IndexVar i, j, k;

    taco::Tensor<float> sparseC("tb_sparseC", {dims[0], dims[0]}, taco::CSR);
    sparseC(i, k) = tb_denseA(i, j) * tb_denseB(j, k);
    sparseC.compile();
    sparseC.assemble();
    sparseC.compute();

    // Remove 0's or else the GPU stuff doesn't work due to mismatches in allocs.
    auto sparseC_no_zeros = sparseC.removeExplicitZeros(taco::CSR);

    int * rowptrC;
    int * colidxC;
    float * valsC;

    taco::getCSRArrays(sparseC_no_zeros, &rowptrC, &colidxC, &valsC);

    auto storage = sparseC_no_zeros.getStorage();
    auto index = storage.getIndex();

    auto colidxArr = index.getModeIndex(1).getIndexArray(1);
    auto coloffsetsize = colidxArr.getSize();
    auto valsCsize = coloffsetsize;

    float * valsC_all_ones = (float *) malloc(valsCsize * sizeof(float));
    for(int z = 0; z < valsCsize; z++){
        valsC_all_ones[z] = 1.0f;
    }

    taco::Tensor<float> sparseC_no_zeros_sparsity_pattern = taco::makeCSR("tb_sparseC_sp_patt", {dims[0], dims[0]},
                                                                          rowptrC, colidxC, valsC_all_ones);

    // Compute result
    taco::IndexVar i_, j_, k_;
    taco::Tensor<float> expected("expected", {dims[0], dims[0]}, taco::CSR);
    expected(i_,j_) = sparseC_no_zeros(i_,j_) * tb_denseA(i_,k_) * tb_denseB(k_,j_);
    auto stmt = expected.getAssignment().concretize();
    stmt = stmt.reorder({i_, j_, k_});
    expected.compile(stmt);
    expected.assemble();
    expected.compute();

    // Get final output values
    auto vals_expected = (float *) expected.getStorage().getValues().getData();

    // std::cout << tb_denseA << std::endl;
    // std::cout << tb_denseB << std::endl;
    // std::cout << sparseC << std::endl;
    // std::cout << sparseC_no_zeros_sparsity_pattern << std::endl;
    // std::cout << expected << std::endl;

    // Host problem definition
    int   A_num_rows   = dims[0];
    int   A_num_cols   = dims[1];
    int   B_num_rows   = dims[1];
    int   B_num_cols   = dims[0];
    int   C_nnz        = valsCsize;
    int   lda          = A_num_cols;
    int   ldb          = B_num_cols;
    int   A_size       = lda * A_num_rows;
    int   B_size       = ldb * B_num_rows;
    float *hA          = valsA;
    float *hB          = valsB;
    int   *hC_offsets  = rowptrC;
    int   *hC_columns  = colidxC;
    float *hC_values   = valsC;
    float *hC_result   = vals_expected;

    thrust::device_vector<float> dC_values_mul(valsC, valsC + valsCsize);
    // float hA[]         = { 1.0f,   2.0f,  3.0f,  4.0f,
    //                     5.0f,   6.0f,  7.0f,  8.0f,
    //                     9.0f,  10.0f, 11.0f, 12.0f,
    //                     13.0f, 14.0f, 15.0f, 16.0f };
    // float hB[]         = {  1.0f,  2.0f,  3.0f,
    //                         4.0f,  5.0f,  6.0f,
    //                         7.0f,  8.0f,  9.0f,
    //                     10.0f, 11.0f, 12.0f };
    // int   hC_offsets[] = { 0, 3, 4, 7, 9 };
    // int   hC_columns[] = { 0, 1, 2, 1, 0, 1, 2, 0, 2 };
    // float hC_values[]  = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f,
    //                     0.0f, 0.0f, 0.0f, 0.0f };
    // float hC_result[]  = { 70.0f, 80.0f, 90.0f,
    //                     184.0f,
    //                     246.0f, 288.0f, 330.0f,
    //                     334.0f, 450.0f };

    float alpha        = 1.0f;
    float beta         = 0.0f;
    //--------------------------------------------------------------------------
    // Device memory management
    int   *dC_offsets, *dC_columns;
    float *dC_values, *dB, *dA;
    CHECK_CUDA( hipMalloc((void**) &dA, A_size * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dB, B_size * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dC_offsets,
                        (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dC_columns, C_nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dC_values,  C_nnz * sizeof(float)) )

    CHECK_CUDA( hipMemcpy(dA, hA, A_size * sizeof(float),
                        hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB, hB, B_size * sizeof(float),
                        hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC_offsets, hC_offsets,
                        (A_num_rows + 1) * sizeof(int),
                        hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC_columns, hC_columns, C_nnz * sizeof(int),
                        hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dC_values, hC_values, C_nnz * sizeof(float),
                        hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseDnMatDescr_t matA, matB;
    hipsparseSpMatDescr_t matC;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create dense matrix A
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matA, A_num_rows, A_num_cols, lda, dA,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )
    // Create dense matrix B
    CHECK_CUSPARSE( hipsparseCreateDnMat(&matB, A_num_cols, B_num_cols, ldb, dB,
                                        HIP_R_32F, HIPSPARSE_ORDER_ROW) )
    // Create sparse matrix C in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matC, A_num_rows, B_num_cols, C_nnz,
                                    dC_offsets, dC_columns, dC_values,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSDDMM_bufferSize(
                                handle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                HIPSPARSE_SDDMM_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute preprocess (optional)
    CHECK_CUSPARSE( hipsparseSDDMM_preprocess(
                                handle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer) )
    // execute SpMM
    CHECK_CUSPARSE( hipsparseSDDMM(handle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                &alpha, matA, matB, &beta, matC, HIP_R_32F,
                                HIPSPARSE_SDDMM_ALG_DEFAULT, dBuffer) )
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )

    thrust::device_vector<float> dC_values_thrust(dC_values, dC_values + valsCsize);
    //--------------------------------------------------------------------------

    // Create output vector
    thrust::device_vector<float> final_out(valsCsize);
    // device result check
    thrust::transform(dC_values_mul.begin(), dC_values_mul.end(), dC_values_thrust.begin(), dC_values_thrust.begin(),
                  thrust::multiplies<float>());

    // CHECK_CUDA( hipMemcpy(hC_values, dC_values, C_nnz * sizeof(float),
    //                     hipMemcpyDeviceToHost) )
    thrust::host_vector<float> final_out_local = dC_values_thrust;

    int correct = 1;
    for (int i = 0; i < C_nnz; i++) {
        if (!float_compare(final_out_local[i], hC_result[i], 0.01f)) {
            std::cout << "VALS GPU: " << final_out_local[i] << " COMPARED TO CPU: " << hC_result[i] << std::endl;
            correct = 0; // direct floating point comparison is not reliable
            break;
        }
    }
    if (correct)
        printf("sddmm_csr_example test PASSED\n");
    else
        printf("sddmm_csr_example test FAILED: wrong result\n");
    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA) )
    CHECK_CUDA( hipFree(dB) )
    CHECK_CUDA( hipFree(dC_offsets) )
    CHECK_CUDA( hipFree(dC_columns) )
    CHECK_CUDA( hipFree(dC_values) )
    return EXIT_SUCCESS;
}

int spmv(taco::Tensor<float> tensorA, taco::Tensor<float> tensorB_csr, benchmark::State * state){

    bool benching = (state != nullptr);

    if(benching) {state->PauseTiming();}

    std::cout << "Calculating SPMv" << std::endl;
    auto dims = tensorA.getDimensions();

    // Get A and its arrays to analyze sizes/etc
    int * rowptrA;
    int * colidxA;
    float * valsA;

    taco::getCSRArrays(tensorA, &rowptrA, &colidxA, &valsA);

    auto storage = tensorA.getStorage();
    auto index = storage.getIndex();
    auto rowptrArr = index.getModeIndex(1).getIndexArray(0);
    auto colidxArr = index.getModeIndex(1).getIndexArray(1);
    auto rowptrsize = rowptrArr.getSize();
    auto numrowsA = rowptrsize - 1;
    auto coloffsetsize = colidxArr.getSize();
    auto valssize = coloffsetsize;
    // Create the float version
    taco::Tensor<float> tensorB = tensorB_csr.transpose("tensorB_dense", {0}, taco::Dense);

    // Create random vector and zero vector
    float * zeroVec = (float *) malloc(dims[0] * sizeof(float));
    for(auto i = 0; i < dims[0]; i++){
        zeroVec[i] = 0.0f;
    }

    taco::IndexVar i, j;
    // Use taco to compute result
    taco::Tensor<float> expected("expected", {dims[0]}, {taco::Dense});
    expected(i) = tensorA(i, j) * tensorB(j);
    expected.compile();
    expected.assemble();
    expected.compute();

    auto vals_expected = (float *) expected.getStorage().getValues().getData();

    auto vec_vals = (float *) tensorB.getStorage().getValues().getData();

   // Host problem definition
    const int A_num_rows      = dims[0];
    const int A_num_cols      = dims[1];
    const int A_nnz           = valssize;
    int       *hA_csrOffsets = rowptrA;
    int       *hA_columns    = colidxA;
    float     *hA_values     = valsA;
    float     *hX            = vec_vals;
    float     *hY            = zeroVec;
    float     *hY_result     = vals_expected;

    float     alpha           = 1.0f;
    float     beta            = 0.0f;
    //--------------------------------------------------------------------------
    // Device memory management
    int   *dA_csrOffsets, *dA_columns;
    float *dA_values, *dX, *dY;
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, A_nnz * sizeof(int))        )
    CHECK_CUDA( hipMalloc((void**) &dA_values,  A_nnz * sizeof(float))      )
    CHECK_CUDA( hipMalloc((void**) &dX,         A_num_cols * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dY,         A_num_rows * sizeof(float)) )

    CHECK_CUDA( hipMemcpy(dA_csrOffsets, hA_csrOffsets,
                           (A_num_rows + 1) * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, hA_values, A_nnz * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dX, hX, A_num_cols * sizeof(float),
                           hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dY, hY, A_num_rows * sizeof(float),
                           hipMemcpyHostToDevice) )
    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;
    void*                dBuffer    = NULL;
    size_t               bufferSize = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                      dA_csrOffsets, dA_columns, dA_values,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    // Create dense vector X
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecX, A_num_cols, dX, HIP_R_32F) )
    // Create dense vector y
    CHECK_CUSPARSE( hipsparseCreateDnVec(&vecY, A_num_rows, dY, HIP_R_32F) )

    if(benching) {state->ResumeTiming();}

    // allocate an external buffer if needed
    CHECK_CUSPARSE( hipsparseSpMV_bufferSize(
                                 handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize) )
    CHECK_CUDA( hipMalloc(&dBuffer, bufferSize) )

    // execute SpMV
    CHECK_CUSPARSE( hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                 &alpha, matA, vecX, &beta, vecY, HIP_R_32F,
                                 HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer) )

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecX) )
    CHECK_CUSPARSE( hipsparseDestroyDnVec(vecY) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    //--------------------------------------------------------------------------
    // device result check
    CHECK_CUDA( hipMemcpy(hY, dY, A_num_rows * sizeof(float),
                           hipMemcpyDeviceToHost) )

    if(benching) {state->PauseTiming();}

    int correct = 1;
    for (int i = 0; i < A_num_rows; i++) {
        if (!float_compare(hY[i], hY_result[i], 0.01f)) { // direct floating point comparison is not
            correct = 0;             // reliable
            break;
        }
    }
    if (correct)
        printf("spmv_csr_example test PASSED\n");
    else
        printf("spmv_csr_example test FAILED: wrong result\n");
    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer) )
    CHECK_CUDA( hipFree(dA_csrOffsets) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    CHECK_CUDA( hipFree(dX) )
    CHECK_CUDA( hipFree(dY) )
    return EXIT_SUCCESS;

}

int mmadd(std::string mat_path){
    std::cout << "Calculating MMADD" << std::endl;
    taco::TensorBase tb = taco::readMTX(mat_path, taco::CSR);
    auto dims = tb.getDimensions();

    // Get A and its arrays to analyze sizes/etc
    int * rowptrA;
    int * colidxA;
    double * valsA_pre;

    taco::getCSRArrays(tb, &rowptrA, &colidxA, &valsA_pre);

    auto storage = tb.getStorage();
    auto index = storage.getIndex();
    auto rowptrArr = index.getModeIndex(1).getIndexArray(0);
    auto colidxArr = index.getModeIndex(1).getIndexArray(1);
    auto rowptrsize = rowptrArr.getSize();
    auto numrowsA = rowptrsize - 1;
    auto coloffsetsize = colidxArr.getSize();
    auto valssize = coloffsetsize;

    float * valsA = (float *) malloc(valssize * sizeof(float));
    for(int i_ = 0; i_ < valssize; i_++){
        valsA[i_] = (float) valsA_pre[i_];
    }

    // Create the float version
    taco::Tensor<float> tb_floatA = taco::makeCSR("tb_float", dims, rowptrA, colidxA, valsA);
    taco::Tensor<float> tb_floatB = tb_floatA.transpose("tb_floatB", {0, 1}, taco::CSR);

    int * rowptrB;
    int * colidxB;
    float * valsB;
    taco::getCSRArrays(tb_floatB, &rowptrB, &colidxB, &valsB);

    int NUM_I = dims[0];
    int NUM_K = dims[1];

    /*
        Compute the output of mmadd
    */
    taco::IndexVar i, k;

    taco::Tensor<float> expected("expected", {NUM_I, NUM_K}, taco::CSR);
    expected(i, k) = tb_floatA(i, k) + tb_floatB(i, k);
    expected.compile();
    expected.assemble();
    expected.compute();

    // Remove 0's or else the GPU stuff doesn't work due to mismatches in allocs.
    auto expected_no_zeros = expected.removeExplicitZeros(taco::CSR);

    // Get the information/ptrs from C for comparison and GPU calls
    int * rowptrC;
    int * colidxC;
    float * valsC;

    taco::getCSRArrays(expected_no_zeros, &rowptrC, &colidxC, &valsC);

    auto storageC = expected_no_zeros.getStorage();
    auto indexC = storageC.getIndex();

    auto rowptrArrC = indexC.getModeIndex(1).getIndexArray(0);
    auto colidxArrC = indexC.getModeIndex(1).getIndexArray(1);

    auto rowptrsizeC = rowptrArrC.getSize();
    auto coloffsetsizeC = colidxArrC.getSize();
    auto valssizeC = coloffsetsizeC;

    // // Host problem definition
    const int A_num_rows = dims[0];
    const int A_num_cols = dims[1];
    const int A_nnz      = valssize;
    const int B_num_rows = dims[1];
    const int B_num_cols = dims[0];
    const int B_nnz      = valssize;

    int  * hA_csrOffsets = rowptrA;
    int  * hA_columns    = colidxA;
    float* hA_values     = valsA;
    int  * hB_csrOffsets = rowptrB;
    int  * hB_columns    = colidxB;
    float* hB_values     = valsB;
    int   *hC_csrOffsets = rowptrC;
    int   *hC_columns    = colidxC;
    float *hC_values     = valsC;

    // const int C_nnz       = valssizeC;
    // #define   C_NUM_NNZ valssizeC   // C compatibility
    float               alpha       = 1.0f;
    float               beta        = 1.0f;
    // hipsparseOperation_t opA         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    // hipsparseOperation_t opB         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    // hipDataType        computeType = HIP_R_32F;
    // //--------------------------------------------------------------------------
    // // Device memory management: Allocate and copy A, B
    int   *dA_csrOffsets, *dA_columns, *dB_csrOffsets, *dB_columns,
        *dC_csrOffsets, *dC_columns;
    float *dA_values, *dB_values, *dC_values;
    // allocate A
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                        (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, A_nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dA_values,  A_nnz * sizeof(float)) )
    // allocate B
    CHECK_CUDA( hipMalloc((void**) &dB_csrOffsets,
                        (B_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_columns, B_nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dB_values,  B_nnz * sizeof(float)) )
    // allocate C offsets
    CHECK_CUDA( hipMalloc((void**) &dC_csrOffsets,
                        (A_num_rows + 1) * sizeof(int)) )
    // copy A
    CHECK_CUDA( hipMemcpy(dA_csrOffsets, hA_csrOffsets,
                        (A_num_rows + 1) * sizeof(int),
                        hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int),
                        hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, hA_values,
                        A_nnz * sizeof(float), hipMemcpyHostToDevice) )
    // copy B
    CHECK_CUDA( hipMemcpy(dB_csrOffsets, hB_csrOffsets,
                        (B_num_rows + 1) * sizeof(int),
                        hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB_columns, hB_columns, B_nnz * sizeof(int),
                        hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB_values, hB_values,
                        B_nnz * sizeof(float), hipMemcpyHostToDevice) )
    // //--------------------------------------------------------------------------
    // // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA, matB, matC;
    hipsparseMatDescr_t matA_nosp, matB_nosp, matC_nosp;
    void*  dBuffer1    = NULL, *dBuffer2   = NULL;
    size_t bufferSize1 = 0,    bufferSize2 = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format

    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                    dA_csrOffsets, dA_columns, dA_values,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matB, B_num_rows, B_num_cols, B_nnz,
                                    dB_csrOffsets, dB_columns, dB_values,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matC, A_num_rows, B_num_cols, 0,
                                    dC_csrOffsets, NULL, NULL,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )

    // Create mat descriptor
    CHECK_CUSPARSE( hipsparseCreateMatDescr( &matA_nosp) )
    CHECK_CUSPARSE( hipsparseCreateMatDescr( &matB_nosp) )
    CHECK_CUSPARSE( hipsparseCreateMatDescr( &matC_nosp) )

    CHECK_CUSPARSE( hipsparseScsrgeam2_bufferSizeExt(handle, dims[0], dims[1],
                                                    &alpha,
                                                    matA_nosp, valssize,
                                                    dA_values, dA_csrOffsets, dA_columns,
                                                    &beta,
                                                    matB_nosp, valssize,
                                                    dB_values, dB_csrOffsets, dB_columns,
                                                    matC_nosp,
                                                    dC_values, dC_csrOffsets, dC_columns,
                                                    &bufferSize1) )

    std::cout << "This many in outputs for C " << bufferSize1 << std::endl;
    CHECK_CUDA( hipMalloc((void**) &dBuffer1, bufferSize1) )

    int nnzC;
    int *nnzptr = &nnzC;

    hipsparseXcsrgeam2Nnz(handle, dims[0], dims[1],
            matA_nosp, valssize, dA_csrOffsets, dA_columns,
            matB_nosp, valssize, dB_csrOffsets, dB_columns,
            matC_nosp, dC_csrOffsets, nnzptr,
            dBuffer1);

    nnzC = *nnzptr;

    std::cout << "Number nonzeros in output according to cuda land..." << nnzC << std::endl;

    // // allocate matrix C
    CHECK_CUDA( hipMalloc((void**) &dC_columns, nnzC * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dC_values,  nnzC * sizeof(float)) )

    hipsparseScsrgeam2(handle, dims[0], dims[1],
            &alpha,
            matA_nosp, valssize,
            dA_values, dA_csrOffsets, dA_columns,
            &beta,
            matB_nosp, valssize,
            dB_values, dB_csrOffsets, dB_columns,
            matC_nosp,
            dC_values, dC_csrOffsets, dC_columns,
            dBuffer1);

    // // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    // //--------------------------------------------------------------------------
    // // device result check
    int   hC_csrOffsets_tmp[dims[0] + 1];
    int   hC_columns_tmp[nnzC];
    float hC_values_tmp[nnzC];
    CHECK_CUDA( hipMemcpy(hC_csrOffsets_tmp, dC_csrOffsets,
                        (A_num_rows + 1) * sizeof(int),
                        hipMemcpyDeviceToHost) )
    CHECK_CUDA( hipMemcpy(hC_columns_tmp, dC_columns, nnzC * sizeof(int),
                        hipMemcpyDeviceToHost) )
    CHECK_CUDA( hipMemcpy(hC_values_tmp, dC_values, nnzC * sizeof(float),
                        hipMemcpyDeviceToHost) )

    int correct = 1;
    for (int i = 0; i < A_num_rows + 1; i++) {
        if (hC_csrOffsets_tmp[i] != hC_csrOffsets[i]) {
            std::cout << "ROWS GPU: " << hC_csrOffsets_tmp[i] << " COMPARED TO CPU: " << hC_csrOffsets[i] << std::endl;
            correct = 0;
            break;
        }
    }

    for (int i = 0; i < nnzC; i++) {
        if (hC_columns_tmp[i] != hC_columns[i]){
            std::cout << "COL GPU: " << hC_columns_tmp[i] << " COMPARED TO CPU: " << hC_columns[i] << std::endl;
            correct = 0;
            break;
        }
        else if(!float_compare(hC_values_tmp[i], hC_values[i], 0.01f)) { // direct floating point
            std::cout << "VAL GPU: " << hC_values_tmp[i] << " COMPARED TO CPU: " << hC_values[i] << std::endl;
            std::cout << fabs(hC_values_tmp[i] - hC_values[i]) << std::endl;
            // std::cout << percent_diff << std::endl;
            correct = 0;                         // comparison is not reliable
            break;
        }
    }
    if (correct)
        printf("spgemm_example test PASSED\n");
    else {
        printf("spgemm_example test FAILED: wrong result\n");
        return EXIT_FAILURE;
    }
    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer1) )
    CHECK_CUDA( hipFree(dBuffer2) )
    CHECK_CUDA( hipFree(dA_csrOffsets) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    CHECK_CUDA( hipFree(dB_csrOffsets) )
    CHECK_CUDA( hipFree(dB_columns) )
    CHECK_CUDA( hipFree(dB_values) )
    CHECK_CUDA( hipFree(dC_csrOffsets) )
    CHECK_CUDA( hipFree(dC_columns) )
    CHECK_CUDA( hipFree(dC_values) )
    return EXIT_SUCCESS;
}

int plus3(taco::Tensor<float> tensorA, taco::Tensor<float> tensorB, taco::Tensor<float> tensorD, benchmark::State * state){

    bool benching = (state != nullptr);

    if(benching) {state->PauseTiming();}

    std::cout << "Calculating PLUS3" << std::endl;
    auto dims = tensorA.getDimensions();

    // Get A and its arrays to analyze sizes/etc
    int * rowptrA;
    int * colidxA;
    float * valsA;

    taco::getCSRArrays(tensorA, &rowptrA, &colidxA, &valsA);

    auto storage = tensorA.getStorage();
    auto index = storage.getIndex();
    auto rowptrArr = index.getModeIndex(1).getIndexArray(0);
    auto colidxArr = index.getModeIndex(1).getIndexArray(1);
    auto rowptrsize = rowptrArr.getSize();
    auto numrowsA = rowptrsize - 1;
    auto coloffsetsize = colidxArr.getSize();
    auto valssize = coloffsetsize;

    int * rowptrB;
    int * colidxB;
    float * valsB;
    taco::getCSRArrays(tensorB, &rowptrB, &colidxB, &valsB);

    int * rowptrD;
    int * colidxD;
    float * valsD;
    taco::getCSRArrays(tensorD, &rowptrD, &colidxD, &valsD);

    int NUM_I = dims[0];
    int NUM_K = dims[1];

    /*
        Compute the output of PLUS3
    */
    taco::IndexVar i, k;

    taco::Tensor<float> expected("expected", {NUM_I, NUM_K}, taco::CSR);
    expected(i, k) = tensorA(i, k) + tensorB(i, k) + tensorD(i, k);
    expected.compile();
    expected.assemble();
    expected.compute();

    // Remove 0's or else the GPU stuff doesn't work due to mismatches in allocs.
    auto expected_no_zeros = expected.removeExplicitZeros(taco::CSR);

    // Get the information/ptrs from C for comparison and GPU calls
    int * rowptrC;
    int * colidxC;
    float * valsC;

    taco::getCSRArrays(expected_no_zeros, &rowptrC, &colidxC, &valsC);

    auto storageC = expected_no_zeros.getStorage();
    auto indexC = storageC.getIndex();

    auto rowptrArrC = indexC.getModeIndex(1).getIndexArray(0);
    auto colidxArrC = indexC.getModeIndex(1).getIndexArray(1);

    auto rowptrsizeC = rowptrArrC.getSize();
    auto coloffsetsizeC = colidxArrC.getSize();
    auto valssizeC = coloffsetsizeC;

    // // Host problem definition
    const int A_num_rows = dims[0];
    const int A_num_cols = dims[1];
    const int A_nnz      = valssize;
    const int B_num_rows = dims[0];
    const int B_num_cols = dims[1];
    const int B_nnz      = valssize;
    const int D_num_rows = dims[0];
    const int D_num_cols = dims[1];
    const int D_nnz      = valssize;

    int  * hA_csrOffsets = rowptrA;
    int  * hA_columns    = colidxA;
    float* hA_values     = valsA;
    int  * hB_csrOffsets = rowptrB;
    int  * hB_columns    = colidxB;
    float* hB_values     = valsB;
    int  * hD_csrOffsets = rowptrD;
    int  * hD_columns    = colidxD;
    float* hD_values     = valsD;
    int   *hC_csrOffsets = rowptrC;
    int   *hC_columns    = colidxC;
    float *hC_values     = valsC;

    float               alpha       = 1.0f;
    float               beta        = 1.0f;
    // //--------------------------------------------------------------------------
    // // Device memory management: Allocate and copy A, B, C, D, E
    int   *dA_csrOffsets, *dA_columns, *dB_csrOffsets, *dB_columns,
        *dC_csrOffsets, *dC_columns, *dD_csrOffsets, *dD_columns, *dE_csrOffsets, *dE_columns;
    float *dA_values, *dB_values, *dC_values, *dD_values, *dE_values;
    // allocate A
    CHECK_CUDA( hipMalloc((void**) &dA_csrOffsets,
                        (A_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dA_columns, A_nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dA_values,  A_nnz * sizeof(float)) )
    // allocate B
    CHECK_CUDA( hipMalloc((void**) &dB_csrOffsets,
                        (B_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_columns, B_nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dB_values,  B_nnz * sizeof(float)) )
    CHECK_CUDA( hipMalloc((void**) &dD_csrOffsets,
                        (D_num_rows + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dD_columns, D_nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dD_values,  D_nnz * sizeof(float)) )
    // allocate C offsets
    CHECK_CUDA( hipMalloc((void**) &dC_csrOffsets,
                        (A_num_rows + 1) * sizeof(int)) )
    // allocate E offsets
    CHECK_CUDA( hipMalloc((void**) &dE_csrOffsets,
                        (A_num_rows + 1) * sizeof(int)) )
    // copy A
    CHECK_CUDA( hipMemcpy(dA_csrOffsets, hA_csrOffsets,
                        (A_num_rows + 1) * sizeof(int),
                        hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_columns, hA_columns, A_nnz * sizeof(int),
                        hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dA_values, hA_values,
    // CHECK_CUDA( hipMemcpy(dA_values, &hA_values,
                        A_nnz * sizeof(float), hipMemcpyHostToDevice) )
    // copy B
    CHECK_CUDA( hipMemcpy(dB_csrOffsets, hB_csrOffsets,
                        (B_num_rows + 1) * sizeof(int),
                        hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB_columns, hB_columns, B_nnz * sizeof(int),
                        hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dB_values, hB_values,
                        B_nnz * sizeof(float), hipMemcpyHostToDevice) )

    // copy D
    CHECK_CUDA( hipMemcpy(dD_csrOffsets, hD_csrOffsets,
                        (D_num_rows + 1) * sizeof(int),
                        hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dD_columns, hD_columns, D_nnz * sizeof(int),
                        hipMemcpyHostToDevice) )
    CHECK_CUDA( hipMemcpy(dD_values, hD_values,
                        D_nnz * sizeof(float), hipMemcpyHostToDevice) )
    // //--------------------------------------------------------------------------
    // // CUSPARSE APIs
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA, matB, matC, matD, matE;
    hipsparseMatDescr_t matA_nosp, matB_nosp, matC_nosp, matD_nosp, matE_nosp;
    void*  dBuffer1    = NULL, *dBuffer2   = NULL;
    size_t bufferSize1 = 0,    bufferSize2 = 0;
    CHECK_CUSPARSE( hipsparseCreate(&handle) )
    // Create sparse matrix A in CSR format

    CHECK_CUSPARSE( hipsparseCreateCsr(&matA, A_num_rows, A_num_cols, A_nnz,
                                    dA_csrOffsets, dA_columns, dA_values,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matB, B_num_rows, B_num_cols, B_nnz,
                                    dB_csrOffsets, dB_columns, dB_values,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matC, A_num_rows, B_num_cols, 0,
                                    dC_csrOffsets, NULL, NULL,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matD, A_num_rows, B_num_cols, 0,
                                    dD_csrOffsets, NULL, NULL,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&matE, A_num_rows, B_num_cols, 0,
                                    dE_csrOffsets, NULL, NULL,
                                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F) )

    // Create mat descriptor
    CHECK_CUSPARSE( hipsparseCreateMatDescr( &matA_nosp) )
    CHECK_CUSPARSE( hipsparseCreateMatDescr( &matB_nosp) )
    CHECK_CUSPARSE( hipsparseCreateMatDescr( &matC_nosp) )
    CHECK_CUSPARSE( hipsparseCreateMatDescr( &matD_nosp) )
    CHECK_CUSPARSE( hipsparseCreateMatDescr( &matE_nosp) )

    if(benching) {state->ResumeTiming();}

    // First computation...C = A + B
    CHECK_CUSPARSE( hipsparseScsrgeam2_bufferSizeExt(handle, dims[0], dims[1],
                                                    &alpha,
                                                    matA_nosp, valssize,
                                                    dA_values, dA_csrOffsets, dA_columns,
                                                    &beta,
                                                    matB_nosp, valssize,
                                                    dB_values, dB_csrOffsets, dB_columns,
                                                    matC_nosp,
                                                    dC_values, dC_csrOffsets, dC_columns,
                                                    &bufferSize1) )

    CHECK_CUDA( hipMalloc((void**) &dBuffer1, bufferSize1) )

    int nnzC;
    int *nnzptr = &nnzC;

    hipsparseXcsrgeam2Nnz(handle, dims[0], dims[1],
            matA_nosp, valssize, dA_csrOffsets, dA_columns,
            matB_nosp, valssize, dB_csrOffsets, dB_columns,
            matC_nosp, dC_csrOffsets, nnzptr,
            dBuffer1);

    nnzC = *nnzptr;

    // // allocate matrix C
    CHECK_CUDA( hipMalloc((void**) &dC_columns, nnzC * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dC_values,  nnzC * sizeof(float)) )

    hipsparseScsrgeam2(handle, dims[0], dims[1],
            &alpha,
            matA_nosp, valssize,
            dA_values, dA_csrOffsets, dA_columns,
            &beta,
            matB_nosp, valssize,
            dB_values, dB_csrOffsets, dB_columns,
            matC_nosp,
            dC_values, dC_csrOffsets, dC_columns,
            dBuffer1);

    // Second computation...E = C + D
    CHECK_CUSPARSE( hipsparseScsrgeam2_bufferSizeExt(handle, dims[0], dims[1],
                                                    &alpha,
                                                    matC_nosp, nnzC,
                                                    dC_values, dC_csrOffsets, dC_columns,
                                                    &beta,
                                                    matD_nosp, valssize,
                                                    dD_values, dD_csrOffsets, dD_columns,
                                                    matE_nosp,
                                                    dE_values, dE_csrOffsets, dE_columns,
                                                    &bufferSize2) )

    CHECK_CUDA( hipMalloc((void**) &dBuffer2, bufferSize2) )

    int nnzE;
    int *nnzptrE = &nnzE;

    hipsparseXcsrgeam2Nnz(handle, dims[0], dims[1],
            matC_nosp, nnzC, dC_csrOffsets, dC_columns,
            matD_nosp, valssize, dD_csrOffsets, dD_columns,
            matE_nosp, dE_csrOffsets, nnzptrE,
            dBuffer2);

    nnzE = *nnzptrE;

    // // allocate matrix C
    CHECK_CUDA( hipMalloc((void**) &dE_columns, nnzE * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dE_values,  nnzE * sizeof(float)) )

    hipsparseScsrgeam2(handle, dims[0], dims[1],
            &alpha,
            matC_nosp, nnzC,
            dC_values, dC_csrOffsets, dC_columns,
            &beta,
            matD_nosp, valssize,
            dD_values, dD_csrOffsets, dD_columns,
            matE_nosp,
            dE_values, dE_csrOffsets, dE_columns,
            dBuffer2);

    // // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseDestroySpMat(matA) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matB) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matC) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matD) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(matE) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
    // //--------------------------------------------------------------------------
    // // device result check
    int   hE_csrOffsets_tmp[dims[0] + 1];
    int   hE_columns_tmp[nnzE];
    float hE_values_tmp[nnzE];
    CHECK_CUDA( hipMemcpy(hE_csrOffsets_tmp, dE_csrOffsets,
                        (A_num_rows + 1) * sizeof(int),
                        hipMemcpyDeviceToHost) )
    CHECK_CUDA( hipMemcpy(hE_columns_tmp, dE_columns, nnzE * sizeof(int),
                        hipMemcpyDeviceToHost) )
    CHECK_CUDA( hipMemcpy(hE_values_tmp, dE_values, nnzE * sizeof(float),
                        hipMemcpyDeviceToHost) )

    if(benching) {state->PauseTiming();}

    taco::Tensor<float> out_gpu = taco::makeCSR("output_from_gpu", {dims[0], dims[1]}, &hE_csrOffsets_tmp[0], &hE_columns_tmp[0], &hE_values_tmp[0]);

    // Need to squeeze out some zeros
    auto out_gpu_no_zeros = out_gpu.removeExplicitZeros(taco::CSR);

    int * rowptr_out;
    int * colidx_out;
    float * vals_out;

    taco::getCSRArrays(out_gpu_no_zeros, &rowptr_out, &colidx_out, &vals_out);

    int correct = 1;
    for (int i = 0; i < A_num_rows + 1; i++) {
        if (rowptr_out[i] != hC_csrOffsets[i]) {
            std::cout << "ROWS GPU: " << rowptr_out[i] << " COMPARED TO CPU: " << hC_csrOffsets[i] << std::endl;
            correct = 0;
            break;
        }
    }

    for (int i = 0; i < nnzE; i++) {
        if (colidx_out[i] != hC_columns[i]){
            std::cout << "COL GPU: " << colidx_out[i] << " COMPARED TO CPU: " << hC_columns[i] << std::endl;
            correct = 0;
            break;
        }
        else if(!float_compare(vals_out[i], hC_values[i], 0.01f)) { // direct floating point
            std::cout << "VAL GPU: " << vals_out[i] << " COMPARED TO CPU: " << hC_values[i] << std::endl;
            std::cout << fabs(vals_out[i] - hC_values[i]) << std::endl;
            correct = 0;                         // comparison is not reliable
            break;
        }
    }
    if (correct)
        printf("plus3 test PASSED\n");
    else {
        printf("plus3 test FAILED: wrong result\n");
        return EXIT_FAILURE;
    }
    //--------------------------------------------------------------------------
    // device memory deallocation
    CHECK_CUDA( hipFree(dBuffer1) )
    CHECK_CUDA( hipFree(dBuffer2) )
    CHECK_CUDA( hipFree(dA_csrOffsets) )
    CHECK_CUDA( hipFree(dA_columns) )
    CHECK_CUDA( hipFree(dA_values) )
    CHECK_CUDA( hipFree(dB_csrOffsets) )
    CHECK_CUDA( hipFree(dB_columns) )
    CHECK_CUDA( hipFree(dB_values) )
    CHECK_CUDA( hipFree(dC_csrOffsets) )
    CHECK_CUDA( hipFree(dC_columns) )
    CHECK_CUDA( hipFree(dC_values) )
    CHECK_CUDA( hipFree(dD_csrOffsets) )
    CHECK_CUDA( hipFree(dD_columns) )
    CHECK_CUDA( hipFree(dD_values) )
    CHECK_CUDA( hipFree(dE_csrOffsets) )
    CHECK_CUDA( hipFree(dE_columns) )
    CHECK_CUDA( hipFree(dE_values) )
    return EXIT_SUCCESS;
}

TensorInputCache<float> inputCacheFloat;

static void cusparse_benchmark(benchmark::State &state, SuiteSparseOp op, bool gen=true, int fill_value = 0) {

  bool GEN_OTHER = (getEnvVar("GEN") == "ON" && gen);

  // Counters must be present in every run to get reported to the CSV.
  state.counters["dimx"] = 0;
  state.counters["dimy"] = 0;
  state.counters["nnz"] = 0;
  state.counters["other_sparsity1"] = 0;
  state.counters["other_sparsity1"] = 0;

  auto tensorPath = getEnvVar("SUITESPARSE_TENSOR_PATH");
  // std::cout << "Running " << opName(op) << " " << tensorPath << std::endl;
  if (tensorPath == "") {
    state.error_occurred();
    return;
  }

  auto pathSplit = taco::util::split(tensorPath, "/");
  auto filename = pathSplit[pathSplit.size() - 1];
  auto tensorName = taco::util::split(filename, ".")[0];
  state.SetLabel(tensorName);

  taco::Tensor<float> tensorA, tensorB, tensorB_pre, tensorC;
//   try {
//     // taco::Format format = op == MATTRANSMUL ? DCSC : DCSR;
//     // std::tie(ssTensor, otherShifted) = inputCacheFloat.getTensorInput(tensorPath, tensorName, format, true /* countNNZ */,
//                                                                 //  true /* includeThird */, true, false, GEN_OTHER);
//     // std::string mat_path = "/home/max/Documents/SPARSE/GPU/mats/Zhao1/Zhao1.mtx";
//     // std::string mat_path = "/home/max/Documents/SPARSE/GPU/mats/fake/fake.mtx";
//     std::tie(tensorA, tensorB_pre) = inputCacheFloat.getTensorInput(mat_path, Zhao1, taco::CSR,
//                                                                        false, false, false, false, true);
//     tensorB = tensorB_pre.transpose("tensorB_transposed", {1, 0}, taco::CSR);

//   } catch (TacoException &e) {
//     // Counters don't show up in the generated CSV if we used SkipWithError, so
//     // just add in the label that this run is skipped.
//     std::cout << e.what() << std::endl;
//     state.SetLabel(tensorName + "/SKIPPED-FAILED-READ");
//     return;
//   }

//   taco::Tensor<int16_t> denseMat1;
//   taco::Tensor<int16_t> denseMat2;
//   taco::Tensor<int16_t> s1("s1"), s2("s2");
//   s1.insert({}, int16_t(2));
//   s2.insert({}, int16_t(2));
//   if (op == SDDMM) {
//     denseMat1 = Tensor<int16_t>("denseMat1", {DIM0, DIM_EXTRA}, Format({dense, dense}));
//     denseMat2 = Tensor<int16_t>("denseMat2", {DIM_EXTRA, DIM1}, Format({dense, dense}, {1, 0}));

//     // (owhsu) Making this dense matrices of all 1's
//     for (int kk = 0; kk < DIM_EXTRA; kk++) {
//       for (int ii = 0; ii < DIM0; ii++) {
//         denseMat1.insert({ii, kk}, int16_t(1));
//       }
//       for (int jj = 0; jj < DIM1; jj++) {
//         denseMat2.insert({kk, jj}, int16_t(1));
//       }
//     }
//   }
    // tensorPath = "/home/max/Documents/SPARSE/GPU/mats/relat3/relat3.mtx";
    // tensorName = "relat3";
    // tensorPath = "/home/max/Documents/SPARSE/GPU/mats/Zhao1/Zhao1.mtx";
    // tensorName = "Zhao1";

  for (auto _: state) {
    state.PauseTiming();
    switch (op) {
      case SPMV: {
        state.PauseTiming();
        std::tie(tensorA, tensorB_pre) = inputCacheFloat.getTensorInput(tensorPath, tensorName, taco::CSR,
                                                                       false, false, /* Include vec*/true,
                                                                       false, false);
        tensorB = inputCacheFloat.otherVecLastMode;
        int DIM0 = tensorA.getDimensions()[0];
        int DIM1 = tensorA.getDimensions()[1];

        state.counters["dimx"] = DIM0;
        state.counters["dimy"] = DIM1;
        state.counters["nnz"] = inputCacheFloat.nnz;

        state.ResumeTiming();
        spmv(tensorA, tensorB, &state);
        state.PauseTiming();
        break;
      }
      case SPMM: {
        state.PauseTiming();
        std::tie(tensorA, tensorB_pre) = inputCacheFloat.getTensorInput(tensorPath, tensorName, taco::CSR,
                                                                        false, false, false, false, true);
        tensorB = tensorB_pre.transpose("tensorB_transposed_csr", {1, 0}, taco::CSR);

        int DIM0 = tensorA.getDimensions()[0];
        int DIM1 = tensorA.getDimensions()[1];

        state.counters["dimx"] = DIM0;
        state.counters["dimy"] = DIM1;
        state.counters["nnz"] = inputCacheFloat.nnz;
        state.ResumeTiming();
        spgemm(tensorA, tensorB, &state);
        state.PauseTiming();
        // stmt = stmt.assemble(result.getAssignment().getLhs().getTensorVar(), taco::AssembleStrategy::Append);
        break;
      }
      case PLUS3: {
        state.PauseTiming();
        std::tie(tensorA, tensorB) = inputCacheFloat.getTensorInput(tensorPath, tensorName, taco::CSR,
                                                                    false, true, /* Include vec*/false,
                                                                    false, false);
        tensorC = inputCacheFloat.thirdTensor;

        int DIM0 = tensorA.getDimensions()[0];
        int DIM1 = tensorA.getDimensions()[1];

        state.counters["dimx"] = DIM0;
        state.counters["dimy"] = DIM1;
        state.counters["nnz"] = inputCacheFloat.nnz;

        state.ResumeTiming();
        plus3(tensorA, tensorB, tensorC, nullptr);
        state.PauseTiming();
        break;
      }
      default:
        state.SkipWithError("invalid expression");
        return;
    }

    // result.compile(stmt);

    // state.ResumeTiming();
    // result.assemble();
    // result.compute();
    // state.PauseTiming();

    // if (auto validationPath = getValidationOutputPath(); validationPath != "") {
    //   auto key = cpuBenchKey(tensorName, opName(op));
    //   auto outpath = validationPath + key + ".tns";
    //   taco::write(outpath, result.removeExplicitZeros(result.getFormat()));
    // }
    // state.ResumeTiming();

  }
}

TACO_BENCH_ARGS(cusparse_benchmark, spmm_ , SPMM, true);
TACO_BENCH_ARGS(cusparse_benchmark, spmv_ , SPMV, true);
TACO_BENCH_ARGS(cusparse_benchmark, plus3_ , PLUS3, true);


#ifdef BENCH_CUSPARSE

BENCHMARK_MAIN();

#else

int main(int argc, char *argv[]) {

    if(argc > 3){
        std::cout << "BAD ARGS..." << std::endl;
        return 0;
    }

    std::string default_mat = "bcsstk01";

    if(argc > 1){
        default_mat = argv[1];
    }

    int default_op = 1;

    if(argc == 3){
        default_op = std::stoi(argv[2]);
    }

    assert(default_op >= 1);
    // assert((default_op >= 1) && (default_op <= 4));

    // Get mat path
    char * mat_path_base_ptr = std::getenv("MAT_PATH");
    assert(mat_path_base_ptr != NULL);
    std::string mat_path_base = mat_path_base_ptr;

    // Load in the matrix
    std::string mat_path = mat_path_base + "/" + default_mat + "/" + default_mat + ".mtx";
    std::cout << "Using path: " << mat_path << std::endl;

    TensorInputCache<float> inputCache;
    taco::Tensor<float> tensorA, tensorB_pre, tensorB, tensorC;

    // return 0;

    switch(default_op){
        // SPGEMM
        case 1:
            std::tie(tensorA, tensorB_pre) = inputCache.getTensorInput(mat_path, default_mat, taco::CSR,
                                                                       false, false, false, false, true);
            tensorB = tensorB_pre.transpose("tensorB_transposed_csr", {1, 0}, taco::CSR);
            return spgemm(tensorA, tensorB, nullptr);
        // SDDMM
        case 2:
            return sddmm(mat_path);
        // SPMV
        case 3:
            std::tie(tensorA, tensorB_pre) = inputCache.getTensorInput(mat_path, default_mat, taco::CSR,
                                                                       false, false, /* Include vec*/true,
                                                                       false, false);
            tensorB = inputCache.otherVecLastMode;

            return spmv(tensorA, tensorB, nullptr);
        // MMADD
        case 4:
            return mmadd(mat_path);
        // PLUS3
        case 5:
            std::tie(tensorA, tensorB) = inputCache.getTensorInput(mat_path, default_mat, taco::CSR,
                                                                       false, true, /* Include vec*/false,
                                                                       false, true);
            tensorC = inputCache.thirdTensor;
            // std::cout << tensorC << std::endl;
            return plus3(tensorA, tensorB, tensorC, nullptr);
        default:
            std::cout << "Invalid OP selected..." << std::endl;
    }

    return 0;

}

#endif